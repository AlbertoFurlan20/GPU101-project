#include "hip/hip_runtime.h"
#include "cuda_header.cuh"

__global__ void convolution2D_for_streams(const float* input, const float* kernel, float* output,
                                    std::pair<int, int> inputSize, std::pair<int, int> filterParams)
{
    extern __shared__ float sharedIndexes[];

    int kernelRadius = filterParams.second;
    int kernelSize = filterParams.first;
    int width = inputSize.first;
    int height = inputSize.second;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int sharedX = threadIdx.x + kernelRadius;
    int sharedY = threadIdx.y + kernelRadius;

    int inputIndex = IDX_2D(x, y, width);

    // Load data into shared memory
    if (x < width && y < height)
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + sharedX] = input[inputIndex];
    }
    else
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + sharedX] = 0.0f;
    }

    // Load halo regions
    if (threadIdx.x < kernelRadius)
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + (sharedX - kernelRadius)] =
            (x >= kernelRadius) ? input[IDX_2D(x - kernelRadius, y, width)] : 0.0f;
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + (sharedX + blockDim.x)] =
            (x + blockDim.x < width) ? input[IDX_2D(x + blockDim.x, y, width)] : 0.0f;
    }

    if (threadIdx.y < kernelRadius)
    {
        sharedIndexes[(sharedY - kernelRadius) * (blockDim.x + 2 * kernelRadius) + sharedX] =
            (y >= kernelRadius) ? input[IDX_2D(x, y - kernelRadius, width)] : 0.0f;
        sharedIndexes[(sharedY + blockDim.y) * (blockDim.x + 2 * kernelRadius) + sharedX] =
            (y + blockDim.y < height) ? input[IDX_2D(x, y + blockDim.y, width)] : 0.0f;
    }

    __syncthreads();

    // Apply convolution
    if (x < width && y < height)
    {
        float partialResult = 0.0f;

        for (int idxY = 0; idxY < kernelSize; ++idxY)
        {
            for (int idxX = 0; idxX < kernelSize; ++idxX)
            {
                int sharedInputY = sharedY - kernelRadius + idxY;
                int sharedInputX = sharedX - kernelRadius + idxX;
                partialResult += kernel[idxY * kernelSize + idxX] *
                    sharedIndexes[sharedInputY * (blockDim.x + 2 * kernelRadius) + sharedInputX];
            }
        }

        output[IDX_2D(x, y, width)] = partialResult;
    }
}

void setup_streams(const float* input, const float* kernel, float* output,
                                  std::pair<int, int> inputSize, std::pair<int, int> filterParams,
                                  int numStreams)
{
    int width = inputSize.first;
    int height = inputSize.second;

    int streamHeight = (height + numStreams - 1) / numStreams;
    hipStream_t* streams = new hipStream_t[numStreams];

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
        int startRow = i * streamHeight;
        int endRow = min((i + 1) * streamHeight, height);

        if (startRow >= height) break;

        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (endRow - startRow + blockDim.y - 1) / blockDim.y);

        convolution2D_for_streams<<<gridDim, blockDim, (blockDim.x + 2 * filterParams.second) * (blockDim.y + 2 * filterParams
            .
            second) * sizeof(float), streams[i]>>>(
            input + startRow * width, kernel, output + startRow * width,
            {width, endRow - startRow}, filterParams);
    }

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    delete[] streams;
}

int main_streams(int dim, float* input, float* filter)
{
    int width = dim;
    int height = dim;

    float* h_input = input;
    float* h_filter = filter;
    float* h_output_streams = new float[width * height];

    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, width * height * sizeof(float));
    hipMalloc(&d_filter, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMalloc(&d_output, width * height * sizeof(float));

    hipMemcpy(d_input, h_input, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, FILTER_SIZE * FILTER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    setup_streams(d_input, d_filter, d_output, {width, height}, {FILTER_SIZE, FILTER_RADIUS}, 4);
    hipMemcpy(h_output_streams, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "[ ";
    for (int i = 0; i < 10; ++i) std::cout << h_output_streams[i] << " ";
    std::cout << "]\n";

    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    delete[] h_output_streams;

    return 0;
}
