#include "hip/hip_runtime.h"
#include "cuda_implementation.cuh"

#include <variant>

using input_type = float;
using filter_type = input_type;

#define TILE_WIDTH 16;
#define TILE_HEIGHT 16;

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)

#define IDX_2D(x, y, width) ((y) * (width) + (x))

template <typename T>
class DynamicArray
{
private:
    T* data;
    size_t size_;

public:
    // Constructor
    DynamicArray(size_t size) : size_(size)
    {
        data = new T[size];
    }

    // Destructor
    ~DynamicArray()
    {
        delete[] data;
    }

    // Prevent copying
    DynamicArray(const DynamicArray&) = delete;
    DynamicArray& operator=(const DynamicArray&) = delete;

    // Allow moving
    DynamicArray(DynamicArray&& other) noexcept
        : data(other.data), size_(other.size_)
    {
        other.data = nullptr;
        other.size_ = 0;
    }

    // Access operators
    T& operator[](size_t index) { return data[index]; }
    const T& operator[](size_t index) const { return data[index]; }

    // Accessors
    size_t size() const { return size_; }
    T* getData() { return data; }
    const T* getData() const { return data; }

    // Initialize data
    void init()
    {
        for (size_t i = 0; i < size_; ++i)
            data[i] = static_cast<T>(rand()) / RAND_MAX;
    }
};

template <typename T>
void printDynamicArray(DynamicArray<T>* array)
{
    int count = 1;
    for (size_t i = 0; i < array->size(); ++i)
    {
        std::cout << array->operator[](i) << " ";

        if (count % 3 == 0) std::cout << "\n";
        if (count == 9)
        {
            std::cout << "\n clamped output\n";
            break;
        }
        count++;
    }

    std::cout << std::endl << std::endl;
}

void checkCudaErrors(hipError_t err, const char* file = __FILE__, int line = __LINE__)
{
    if (err != hipSuccess)
    {
        std::cerr << "\nCUDA Error at: " << file << ":" << line << std::endl
            << "* Error code: " << static_cast<int>(err) << std::endl
            << "* Error type: " << hipGetErrorName(err) << std::endl
            << "* Error description: " << hipGetErrorString(err) << std::endl;

        // Get last error state
        hipError_t lastError = hipGetLastError();
        if (lastError != err)
        {
            std::cerr << "Additional last error: " << hipGetErrorString(lastError) << std::endl;
        }

        // Ensure all previous operations have completed
        hipDeviceSynchronize();

        // Reset device to clear any errors
        hipDeviceReset();

        exit(EXIT_FAILURE);
    }
}

__global__ void singleKernelConvolution2D(const float* input, const float* kernel, float* output,
                                          std::pair<int, int> inputSize, std::pair<int, int> filterParams)
{
    extern __shared__ float sharedIndexes[];

    int kernelRadius = filterParams.second;
    int kernelSize = filterParams.first;
    int width = inputSize.first;
    int height = inputSize.second;

    int tileWidth = blockDim.x; // Width of the tile
    int tileHeight = blockDim.y; // Height of the tile

    int x = blockIdx.x * tileWidth + threadIdx.x; // Global x index
    int y = blockIdx.y * tileHeight + threadIdx.y; // Global y index

    int sharedX = threadIdx.x + kernelRadius; // Offset for shared memory
    int sharedY = threadIdx.y + kernelRadius;

    // Load data into shared memory
    if (x < width && y < height)
    {
        sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + sharedX] = input[IDX_2D(x, y, width)];
    }
    else
    {
        sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
    }

    // Load halo regions (neighboring pixels outside the tile)
    if (threadIdx.x < kernelRadius)
    {
        if (x >= kernelRadius)
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX - kernelRadius)] =
                input[IDX_2D(x - kernelRadius, y, width)];
        }
        else
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX - kernelRadius)] = 0.0f;
        }

        if (x + tileWidth < width)
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX + tileWidth)] =
                input[IDX_2D(x + tileWidth, y, width)];
        }
        else
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX + tileWidth)] = 0.0f;
        }
    }

    if (threadIdx.y < kernelRadius)
    {
        if (y >= kernelRadius)
        {
            sharedIndexes[(sharedY - kernelRadius) * (tileWidth + 2 * kernelRadius) + sharedX] =
                input[IDX_2D(x, y - kernelRadius, width)];
        }
        else
        {
            sharedIndexes[(sharedY - kernelRadius) * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
        }

        if (y + tileHeight < height)
        {
            sharedIndexes[(sharedY + tileHeight) * (tileWidth + 2 * kernelRadius) + sharedX] =
                input[IDX_2D(x, y + tileHeight, width)];
        }
        else
        {
            sharedIndexes[(sharedY + tileHeight) * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
        }
    }

    __syncthreads(); // Synchronize threads to ensure all data is loaded

    // Apply convolution
    if (x < width && y < height)
    {
        float partialResult = 0.0f;

        for (int idxY = 0; idxY < kernelSize; ++idxY)
        {
            for (int idxX = 0; idxX < kernelSize; ++idxX)
            {
                int sharedInputY = sharedY - kernelRadius + idxY;
                int sharedInputX = sharedX - kernelRadius + idxX;
                partialResult += kernel[idxY * kernelSize + idxX] *
                    sharedIndexes[sharedInputY * (tileWidth + 2 * kernelRadius) + sharedInputX];
            }
        }

        output[IDX_2D(x, y, width)] = partialResult;
    }
}

__global__ void multiKernelConvolution2D(float* input, float* output, int inputWidth, int inputHeight, float* filter,
                                         int filterSize, int sharedMemSize, int startX, int startY)
{
    // Declare extern shared memory
    extern __shared__ float sharedMem[];

    // Calculate the thread's position in the block
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // Shared memory indexing (linearized for 2D access)
    int sharedX = threadIdx.x + FILTER_RADIUS;
    int sharedY = threadIdx.y + FILTER_RADIUS;

    // Load data into shared memory (considering halo regions)
    if (tx < inputWidth && ty < inputHeight)
    {
        int globalX = tx + startX;
        int globalY = ty + startY;
        sharedMem[sharedX + sharedY * (blockDim.x + 2 * FILTER_RADIUS)] = input[globalY * inputWidth + globalX];

        // Load data from the edges of the tile into shared memory (halo region)
        if (threadIdx.x < FILTER_RADIUS && globalX - FILTER_RADIUS >= 0)
        {
            sharedMem[(sharedX - FILTER_RADIUS) + sharedY * (blockDim.x + 2 * FILTER_RADIUS)] = input[globalY *
                inputWidth + (globalX - FILTER_RADIUS)];
        }
        if (threadIdx.x >= blockDim.x - FILTER_RADIUS && globalX + FILTER_RADIUS < inputWidth)
        {
            sharedMem[(sharedX + FILTER_RADIUS) + sharedY * (blockDim.x + 2 * FILTER_RADIUS)] = input[globalY *
                inputWidth + (globalX + FILTER_RADIUS)];
        }
        if (threadIdx.y < FILTER_RADIUS && globalY - FILTER_RADIUS >= 0)
        {
            sharedMem[sharedX + (sharedY - FILTER_RADIUS) * (blockDim.x + 2 * FILTER_RADIUS)] = input[(globalY -
                FILTER_RADIUS) * inputWidth + globalX];
        }
        if (threadIdx.y >= blockDim.y - FILTER_RADIUS && globalY + FILTER_RADIUS < inputHeight)
        {
            sharedMem[sharedX + (sharedY + FILTER_RADIUS) * (blockDim.x + 2 * FILTER_RADIUS)] = input[(globalY +
                FILTER_RADIUS) * inputWidth + globalX];
        }
    }

    __syncthreads(); // Synchronize to ensure all threads have loaded data into shared memory

    // Apply convolution (only for valid pixels within the current tile)
    if (tx < inputWidth && ty < inputHeight)
    {
        float result = 0.0f;
        for (int fy = -FILTER_RADIUS; fy <= FILTER_RADIUS; fy++)
        {
            for (int fx = -FILTER_RADIUS; fx <= FILTER_RADIUS; fx++)
            {
                result += sharedMem[(sharedX + fx) + (sharedY + fy) * (blockDim.x + 2 * FILTER_RADIUS)] * filter[(fy +
                    FILTER_RADIUS) * filterSize + (fx + FILTER_RADIUS)];
            }
        }
        int globalIdx = (ty + startY) * inputWidth + (tx + startX);
        if (globalIdx < inputWidth * inputHeight)
        {
            output[globalIdx] = result;
        }
    }
}

std::pair<dim3, dim3> setSizeAndGrid(int convolutionType, std::pair<int, int> inputParams)
{
    dim3 blockDim(1, 1, 1);
    dim3 gridSize(1, 1, 1);

    int width = inputParams.first;
    int height = inputParams.second;

    // Query device properties for block and grid size constraints
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int maxBlockX = deviceProp.maxThreadsDim[0];
    int maxBlockY = deviceProp.maxThreadsDim[1];
    int maxGridX = deviceProp.maxGridSize[0];
    int maxGridY = deviceProp.maxGridSize[1];
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    if (convolutionType == 2)
    {
        // Choose block dimensions (16x16 threads is common for 2D)
        blockDim.x = std::min(16, maxBlockX);
        blockDim.y = std::min(16, maxBlockY);

        // Ensure blockDim does not exceed maxThreadsPerBlock
        if (blockDim.x * blockDim.y > maxThreadsPerBlock)
        {
            blockDim.y = maxThreadsPerBlock / blockDim.x;
        }

        // Calculate grid dimensions
        gridSize.x = (width + blockDim.x - 1) / blockDim.x;
        gridSize.y = (height + blockDim.y - 1) / blockDim.y;

        // Clamp grid dimensions to device constraints
        gridSize.x = std::min(static_cast<int>(gridSize.x), maxGridX);
        gridSize.y = std::min(static_cast<int>(gridSize.y), maxGridY);
    }
    else
    {
        std::cerr << "[WARNING]:: Only 2D convolution is supported!" << std::endl;
    }

    return std::make_pair(gridSize, blockDim);
}

std::tuple<int, int*, int*> getDeviceConstraints()
{
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return std::make_tuple(-1, static_cast<int*>(nullptr), static_cast<int*>(nullptr));
    }

    if (deviceCount > 0)
    {
        // For simplicity, i'll assume 1x device
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        auto gridSizeConstraints = new int[3];
        auto blockConstraints = new int[3];

        for (int i = 0; i < 3; i++)
        {
            gridSizeConstraints[i] = deviceProp.maxGridSize[i];
            blockConstraints[i] = deviceProp.maxThreadsDim[i];
        }

        auto sharedMemoryConstraints = deviceProp.sharedMemPerBlock;

        return std::make_tuple(sharedMemoryConstraints, gridSizeConstraints, blockConstraints);
    }

    return std::make_tuple(-1, static_cast<int*>(nullptr), static_cast<int*>(nullptr));
}

std::tuple<int, int, int> parseInput(int argc, char** argv)
{
    if (argc < 2)
    {
        printf("Please specify:\n- matrix dimensions (1);\n- convolution type (2)");
        std::exit(1);
    }

    unsigned convolutionType;
    const unsigned dim = atoi(argv[1]);

    std::cout << "dim: " << dim << "\n";
    if (argc > 2)
    {
        convolutionType = atoi(argv[2]);
        std::cout << "convolution type: " << convolutionType << "D";
    }
    else
    {
        convolutionType = 2;
        std::cout << "convolution type: " << convolutionType << "D (defaulted)";
    }

    std::cout << "[y] - shared memory\n";

    if (convolutionType < 1 || convolutionType > 3)
    {
        std::cout << "\n[ERROR]:: supported convolution: 2D\n";
        std::exit(1);
    }

    // up to 3D convolution is supported
    assert(convolutionType == 1 || convolutionType == 2 || convolutionType == 3);

    std::cout << "supported convolution: 2D\n";

    return {dim, dim, convolutionType};
}

std::tuple<DynamicArray<float>*, DynamicArray<float>*, DynamicArray<float>*> initArgs(int raw_size)
{
    auto input = new DynamicArray<float>(raw_size);
    auto output = new DynamicArray<float>(raw_size);
    auto filter = new DynamicArray<float>(FILTER_SIZE * FILTER_SIZE);

    filter->init();
    input->init();

    return {input, output, filter};
}

std::tuple<float*, float*, float*> allocateAndInitDeviceMemory(DynamicArray<float>* filter, DynamicArray<float>* input,
                                                               DynamicArray<float>* output)
{
    float *d_filter, *d_input, *d_output;

    checkCudaErrors(hipMalloc(&d_filter, filter->size() * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_input, input->size() * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_output, output->size() * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_input, input->getData(), input->size() * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_filter, filter->getData(), filter->size() * sizeof(float), hipMemcpyHostToDevice));

    return {d_filter, d_input, d_output};
}

bool performComputationalFeasibility(std::tuple<int, int> sharedMemoryParams, std::tuple<int*, dim3> gridSizeParams,
                                     std::tuple<int*, dim3> blockSizeParams)
{
    auto [sharedMax, sharedAmount] = sharedMemoryParams;
    auto [gridMax, gridAmount] = gridSizeParams;
    auto [blockMax, blockAmount] = blockSizeParams;

    // shared memory check
    if (sharedMax < sharedAmount)
    {
        return false;
    }

    if (gridMax[0] < gridAmount.x || gridMax[1] < gridAmount.y || gridMax[2] < gridAmount.y)
    {
        return false;
    }

    if (blockMax[0] < blockAmount.x || blockMax[1] < blockAmount.y || blockMax[2] < blockAmount.y)
    {
        return false;
    }

    return true;
}

std::variant<bool, std::tuple<dim3, dim3, std::pair<int, int>, std::pair<int, int>, int>> runSizeSetup(
    int width, int height, int convolutionType)
{
    auto inputParams = std::make_pair(width, height);
    auto filterParams = std::make_pair(FILTER_SIZE, FILTER_RADIUS);

    auto [gridSize, blockDim] = setSizeAndGrid(convolutionType, inputParams);

    int sharedMemSize = (blockDim.x + 2 * FILTER_RADIUS) * (blockDim.y + 2 * FILTER_RADIUS) * sizeof(float);

    auto [sharedMemConstraint, gridSizeConstraint, blockSizeConstraint] = getDeviceConstraints();

    std::cout << "\n[SETUP]:: set block size :=(" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")";
    std::cout << "          max size:=(" << blockSizeConstraint[0] << ", " << blockSizeConstraint[1] << ", " <<
        blockSizeConstraint[2] << ")";

    std::cout << "\n[SETUP]:: set grid size :=(" << gridSize.x << ", " << gridSize.y << ", " << gridSize.z << ")";
    std::cout << "          max size:=(" << gridSizeConstraint[0] << ", " << gridSizeConstraint[1] << ", " <<
        gridSizeConstraint[2] << ")";

    std::cout << "\n[SETUP]:: set shared mem size:= " << sharedMemSize << std::endl;
    std::cout << "          max shared mem:= " << sharedMemConstraint << std::endl;

    if (performComputationalFeasibility({sharedMemConstraint, sharedMemSize},
                                        {gridSizeConstraint, gridSize},
                                        {blockSizeConstraint, blockDim}))
    {
        return false;
    }

    return std::make_tuple(gridSize, blockDim, inputParams, filterParams, sharedMemSize);
}

template <typename T>
float launchSingleKernel(std::variant<bool, T> outcome, float* d_input, float* d_filter, float* d_output)
{
    auto [gridSize, blockDim, inputParams, filterParams, sharedMemSize] = std::get<std::tuple<
        dim3, dim3, std::pair<int, int>, std::pair<int, int>, int>>(outcome);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // 3. kernel launch
    // Record the start time
    checkCudaErrors(hipEventRecord(start));

    convolution2D<<<gridSize, blockDim, sharedMemSize>>>(d_input, d_filter, d_output, inputParams, filterParams);

    // 4. device synch & mem copy backwards
    checkCudaErrors(hipDeviceSynchronize());

    // Record the stop time
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

    // Calculate and display elapsed time
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    // Cleanup timing events
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}

float launchStream(float* d_input, float* d_filter, float* d_output, std::pair<int, int> inputSize)
{
    auto tileWidth = TILE_WIDTH;
    auto tileHeight = TILE_HEIGHT;

    auto inputWidth = inputSize.first;
    auto inputHeight = inputSize.second;

    // Number of tiles in X and Y directions
    int numTilesX = (inputWidth + tileWidth - 1) / tileWidth;
    int numTilesY = (inputHeight + tileHeight - 1) / tileHeight;

    // Number of streams
    int numStreams = 4;
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
    }

    int streamIdx = 0;
    int sharedMemSize = (tileWidth + 2 * FILTER_RADIUS) * (tileHeight + 2 * FILTER_RADIUS) * sizeof(float);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // 3. kernel launch
    // Record the start time
    checkCudaErrors(hipEventRecord(start));

    // Launch kernels for each tile
    for (int ty = 0; ty < numTilesY; ++ty)
    {
        for (int tx = 0; tx < numTilesX; ++tx)
        {
            // Start position for each tile
            int startX = tx * tileWidth;
            int startY = ty * tileHeight;

            // Kernel grid and block sizes
            dim3 blockSize(tileWidth,tileHeight);
            dim3 gridSize((tileWidth + blockSize.x - 1) / blockSize.x, (tileHeight + blockSize.y - 1) / blockSize.y);

            // Launch convolution kernel using streams
            multiKernelConvolution2D<<<gridSize, blockSize, sharedMemSize, streams[streamIdx]>>>(
                d_input, d_output, inputWidth, inputHeight, d_filter, FILTER_SIZE, sharedMemSize, startX, startY);
            streamIdx = (streamIdx + 1) % numStreams; // Cycle through streams
        }
    }

    // Synchronize all streams
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamSynchronize(streams[i]);
    }

    // Cleanup
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamDestroy(streams[i]);
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

    // Calculate and display elapsed time
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    // Cleanup timing events
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}

int run_assignment_cuda(int argc, char** argv)
{
    auto [width, height, convolutionType] = parseInput(argc, argv);

    auto [input, output, filter] = initArgs(width * height);

    assert(output->size() == input->size());

    // 1. memory allocation
    auto [d_filter, d_input, d_output] = allocateAndInitDeviceMemory(filter, input, output);

    // 2. gridsize & blocksize setup
    auto outcome = runSizeSetup(width, height, convolutionType);

    float milliseconds = 0.0f;

    if (std::holds_alternative<bool>(outcome))
    {
        milliseconds = launchStream(d_input, d_filter, d_output, {width, height});
    }
    else
    {
        milliseconds = launchSingleKernel(outcome, d_input, d_filter, d_output);
    }

    std::cout << "\nKernel execution time: " << milliseconds << " ms\n";

    // Copy result back to host
    checkCudaErrors(hipMemcpy(output->getData(), d_output, output->size() * sizeof(float),
                               hipMemcpyDeviceToHost));

    std::cout << "[OUTPUT]\n";
    printDynamicArray(output);

    // Cleanup and deallocate memory
    delete output;
    delete filter;
    delete input;

    hipFree(d_filter);
    hipFree(d_input);
    hipFree(d_output);

    return EXIT_SUCCESS;
}
