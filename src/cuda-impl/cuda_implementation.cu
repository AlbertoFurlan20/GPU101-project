#include "hip/hip_runtime.h"
#include "cuda_implementation.cuh"

#include <variant>
#include <omp.h> // OpenMP for parallelization
#include <iostream>
#include <random>
#include <vector>

using input_type = float;
using filter_type = input_type;

#define TILE_WIDTH 512
#define TILE_HEIGHT 512

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)

#define IDX_2D(x, y, width) ((y) * (width) + (x))

template <typename T>
class DynamicArray
{
private:
    T* data;
    size_t size_;

public:
    // Constructor
    DynamicArray(size_t size) : size_(size)
    {
        data = new T[size];
    }

    // Destructor
    ~DynamicArray()
    {
        delete[] data;
    }

    // Prevent copying
    DynamicArray(const DynamicArray&) = delete;
    DynamicArray& operator=(const DynamicArray&) = delete;

    // Allow moving
    DynamicArray(DynamicArray&& other) noexcept
        : data(other.data), size_(other.size_)
    {
        other.data = nullptr;
        other.size_ = 0;
    }

    // Access operators
    T& operator[](size_t index) { return data[index]; }
    const T& operator[](size_t index) const { return data[index]; }

    // Accessors
    size_t size() const { return size_; }
    T* getData() { return data; }
    const T* getData() const { return data; }

    // Initialize data
    void init()
    {
        // Mersenne Twister random number generator
        std::random_device rd;
        std::mt19937 mt(rd()); // Use mt19937 for faster random number generation
        std::uniform_real_distribution<T> dist(0.0, 100.0);

        // Parallel initialization using OpenMP
#pragma omp parallel
        {
            // Each thread gets its own RNG to avoid contention
            std::mt19937 thread_rng(mt());

#pragma omp for
            for (size_t i = 0; i < size_; ++i)
            {
                data[i] = dist(thread_rng); // Assign random float in [0.0, 1.0]
            }
        }
    }
};

template <typename T>
void printDynamicArray(DynamicArray<T>* array)
{
    int count = 1;
    for (size_t i = 0; i < array->size(); ++i)
    {
        std::cout << array->operator[](i) << " ";

        if (count % 3 == 0) std::cout << "\n";
        if (count == 9)
        {
            std::cout << "\n clamped output\n";
            break;
        }
        count++;
    }

    std::cout << std::endl << std::endl;
}

void checkCudaErrors(hipError_t err, const char* file = __FILE__, int line = __LINE__)
{
    if (err != hipSuccess)
    {
        std::cerr << "\nCUDA Error at: " << file << ":" << line << std::endl
            << "* Error code: " << static_cast<int>(err) << std::endl
            << "* Error type: " << hipGetErrorName(err) << std::endl
            << "* Error description: " << hipGetErrorString(err) << std::endl;

        // Get last error state
        hipError_t lastError = hipGetLastError();
        if (lastError != err)
        {
            std::cerr << "Additional last error: " << hipGetErrorString(lastError) << std::endl;
        }

        // Ensure all previous operations have completed
        hipDeviceSynchronize();

        // Reset device to clear any errors
        hipDeviceReset();

        exit(EXIT_FAILURE);
    }
}

__global__ void singleKernelConvolution2D(const float* input, const float* kernel, float* output,
                                          std::pair<int, int> inputSize, std::pair<int, int> filterParams)
{
    extern __shared__ float sharedIndexes[];

    int kernelRadius = filterParams.second;
    int kernelSize = filterParams.first;
    int width = inputSize.first;
    int height = inputSize.second;

    int tileWidth = blockDim.x; // Width of the tile
    int tileHeight = blockDim.y; // Height of the tile

    int x = blockIdx.x * tileWidth + threadIdx.x; // Global x index
    int y = blockIdx.y * tileHeight + threadIdx.y; // Global y index

    int sharedX = threadIdx.x + kernelRadius; // Offset for shared memory
    int sharedY = threadIdx.y + kernelRadius;

    // Load data into shared memory
    if (x < width && y < height)
    {
        sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + sharedX] = input[IDX_2D(x, y, width)];
    }
    else
    {
        sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
    }

    // Load halo regions (neighboring pixels outside the tile)
    if (threadIdx.x < kernelRadius)
    {
        if (x >= kernelRadius)
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX - kernelRadius)] =
                input[IDX_2D(x - kernelRadius, y, width)];
        }
        else
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX - kernelRadius)] = 0.0f;
        }

        if (x + tileWidth < width)
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX + tileWidth)] =
                input[IDX_2D(x + tileWidth, y, width)];
        }
        else
        {
            sharedIndexes[sharedY * (tileWidth + 2 * kernelRadius) + (sharedX + tileWidth)] = 0.0f;
        }
    }

    if (threadIdx.y < kernelRadius)
    {
        if (y >= kernelRadius)
        {
            sharedIndexes[(sharedY - kernelRadius) * (tileWidth + 2 * kernelRadius) + sharedX] =
                input[IDX_2D(x, y - kernelRadius, width)];
        }
        else
        {
            sharedIndexes[(sharedY - kernelRadius) * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
        }

        if (y + tileHeight < height)
        {
            sharedIndexes[(sharedY + tileHeight) * (tileWidth + 2 * kernelRadius) + sharedX] =
                input[IDX_2D(x, y + tileHeight, width)];
        }
        else
        {
            sharedIndexes[(sharedY + tileHeight) * (tileWidth + 2 * kernelRadius) + sharedX] = 0.0f;
        }
    }

    __syncthreads(); // Synchronize threads to ensure all data is loaded

    // Apply convolution
    if (x < width && y < height)
    {
        float partialResult = 0.0f;

        for (int idxY = 0; idxY < kernelSize; ++idxY)
        {
            for (int idxX = 0; idxX < kernelSize; ++idxX)
            {
                int sharedInputY = sharedY - kernelRadius + idxY;
                int sharedInputX = sharedX - kernelRadius + idxX;
                partialResult += kernel[idxY * kernelSize + idxX] *
                    sharedIndexes[sharedInputY * (tileWidth + 2 * kernelRadius) + sharedInputX];
            }
        }

        output[IDX_2D(x, y, width)] = partialResult;
    }
}

__global__ void multiKernelConvolution2D(float* input, float* output, int inputWidth, int inputHeight, float* filter,
                                         int filterSize, int startX, int startY)
{
    extern __shared__ float sharedMem[];

    int sharedWidth = blockDim.x + 2 * FILTER_RADIUS;

    // Thread position in the global grid
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // Thread position in shared memory
    int sharedX = threadIdx.x + FILTER_RADIUS;
    int sharedY = threadIdx.y + FILTER_RADIUS;

    // Global indices for this thread
    int globalX = tx + startX;
    int globalY = ty + startY;

    // Initialize shared memory with zeros
    sharedMem[sharedX + sharedY * sharedWidth] = 0.0f;

    // Load the current block into shared memory
    if (globalX < inputWidth && globalY < inputHeight)
    {
        sharedMem[sharedX + sharedY * sharedWidth] = input[IDX_2D(globalX, globalY, inputWidth)];
    }

    // Handle halo regions safely
    if (threadIdx.x < FILTER_RADIUS && (globalX - FILTER_RADIUS) >= 0)
    {
        sharedMem[(sharedX - FILTER_RADIUS) + sharedY * sharedWidth] =
            (globalX - FILTER_RADIUS) < inputWidth && globalY < inputHeight
                ? input[IDX_2D(globalX - FILTER_RADIUS, globalY, inputWidth)]
                : 0.0f;
    }

    if (threadIdx.x >= blockDim.x - FILTER_RADIUS && (globalX + FILTER_RADIUS) < inputWidth)
    {
        sharedMem[(sharedX + FILTER_RADIUS) + sharedY * sharedWidth] =
            (globalX + FILTER_RADIUS) < inputWidth && globalY < inputHeight
                ? input[IDX_2D(globalX + FILTER_RADIUS, globalY, inputWidth)]
                : 0.0f;
    }

    if (threadIdx.y < FILTER_RADIUS && (globalY - FILTER_RADIUS) >= 0)
    {
        sharedMem[sharedX + (sharedY - FILTER_RADIUS) * sharedWidth] =
            globalX < inputWidth && (globalY - FILTER_RADIUS) < inputHeight
                ? input[IDX_2D(globalX, globalY - FILTER_RADIUS, inputWidth)]
                : 0.0f;
    }

    if (threadIdx.y >= blockDim.y - FILTER_RADIUS && (globalY + FILTER_RADIUS) < inputHeight)
    {
        sharedMem[sharedX + (sharedY + FILTER_RADIUS) * sharedWidth] =
            globalX < inputWidth && (globalY + FILTER_RADIUS) < inputHeight
                ? input[IDX_2D(globalX, globalY + FILTER_RADIUS, inputWidth)]
                : 0.0f;
    }

    __syncthreads();

    // Apply convolution if within valid global bounds
    if (globalX < inputWidth && globalY < inputHeight)
    {
        float result = 0.0f;
        for (int fy = -FILTER_RADIUS; fy <= FILTER_RADIUS; fy++)
        {
            for (int fx = -FILTER_RADIUS; fx <= FILTER_RADIUS; fx++)
            {
                int sharedIdx = (sharedX + fx) + (sharedY + fy) * sharedWidth;
                int filterIdx = (fy + FILTER_RADIUS) * filterSize + (fx + FILTER_RADIUS);
                result += sharedMem[sharedIdx] * filter[filterIdx];
            }
        }
        output[IDX_2D(globalX, globalY, inputWidth)] = result;
    }
}

std::pair<dim3, dim3> setSizeAndGrid(int convolutionType, std::pair<int, int> inputParams)
{
    dim3 blockDim(1, 1, 1);
    dim3 gridSize(1, 1, 1);

    int width = inputParams.first;
    int height = inputParams.second;

    // Query device properties for block and grid size constraints
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int maxBlockX = deviceProp.maxThreadsDim[0];
    int maxBlockY = deviceProp.maxThreadsDim[1];
    int maxGridX = deviceProp.maxGridSize[0];
    int maxGridY = deviceProp.maxGridSize[1];
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    if (convolutionType == 2)
    {
        // Choose block dimensions (16x16 threads is common for 2D)
        blockDim.x = std::min(16, maxBlockX);
        blockDim.y = std::min(16, maxBlockY);

        // Ensure blockDim does not exceed maxThreadsPerBlock
        if (blockDim.x * blockDim.y > maxThreadsPerBlock)
        {
            blockDim.y = maxThreadsPerBlock / blockDim.x;
        }

        // Calculate grid dimensions
        gridSize.x = (width + blockDim.x - 1) / blockDim.x;
        gridSize.y = (height + blockDim.y - 1) / blockDim.y;

        // Clamp grid dimensions to device constraints
        gridSize.x = std::min(static_cast<int>(gridSize.x), maxGridX);
        gridSize.y = std::min(static_cast<int>(gridSize.y), maxGridY);
    }
    else
    {
        std::cerr << "[WARNING]:: Only 2D convolution is supported!" << std::endl;
    }

    return std::make_pair(gridSize, blockDim);
}

std::tuple<int, int*, int*> getDeviceConstraints()
{
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return std::make_tuple(-1, static_cast<int*>(nullptr), static_cast<int*>(nullptr));
    }

    if (deviceCount > 0)
    {
        // For simplicity, i'll assume 1x device
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        auto gridSizeConstraints = new int[3];
        auto blockConstraints = new int[3];

        for (int i = 0; i < 3; i++)
        {
            gridSizeConstraints[i] = deviceProp.maxGridSize[i];
            blockConstraints[i] = deviceProp.maxThreadsDim[i];
        }

        auto sharedMemoryConstraints = deviceProp.sharedMemPerBlock;

        return std::make_tuple(sharedMemoryConstraints, gridSizeConstraints, blockConstraints);
    }

    return std::make_tuple(-1, static_cast<int*>(nullptr), static_cast<int*>(nullptr));
}

std::tuple<int, int, int> parseInput(int argc, char** argv)
{
    std::cout << "[INFO]\n";
    if (argc < 2)
    {
        std::cerr << "Please specify:\n- matrix dimensions (1);\n- convolution type (2)" << std::endl;
        std::exit(1);
    }

    int convolutionType;
    int dim;

    try
    {
        // Convert argv[1] to integer (matrix dimensions)
        dim = std::stoi(argv[1]);
        std::cout << " * dim: " << dim << "\n";

        // Convert argv[2] to integer (convolution type) if provided
        if (argc > 2)
        {
            convolutionType = std::stoi(argv[2]);
            std::cout << " * convolution type: " << convolutionType << "D ";
        }
        else
        {
            convolutionType = 2;
            std::cout << " * convolution type: " << convolutionType << "D (defaulted) ";
        }

        // Validate convolution type
        if (convolutionType < 1 || convolutionType > 3)
        {
            throw std::out_of_range("Supported convolution types are 1D, 2D, and 3D.");
        }
    }
    catch (const std::exception& e)
    {
        std::cerr << "Error parsing input: " << e.what() << std::endl;
        std::exit(1);
    }

    std::cout << "[y] - shared memory\n";
    std::cout << " * supported convolution: " << convolutionType << "D\n";

    return std::make_tuple(dim, dim, convolutionType);
}

std::tuple<DynamicArray<float>*, DynamicArray<float>*, DynamicArray<float>*> initArgs(int width, int height)
{
    try
    {
        int prod = width * height;
        auto input = new DynamicArray<float>(prod);
        auto output = new DynamicArray<float>(prod);
        auto filter = new DynamicArray<float>(FILTER_SIZE * FILTER_SIZE);

        filter->init();
        input->init();

        return {input, output, filter};
    }
    catch (std::exception& e)
    {
        long long prod = static_cast<long long>(width) * height;
        auto input = new DynamicArray<float>(prod);
        auto output = new DynamicArray<float>(prod);
        auto filter = new DynamicArray<float>(FILTER_SIZE * FILTER_SIZE);

        filter->init();
        input->init();

        return {input, output, filter};
    }
}

std::tuple<float*, float*, float*> allocateAndInitDeviceMemory(DynamicArray<float>* filter, DynamicArray<float>* input,
                                                               DynamicArray<float>* output)
{
    float *d_filter, *d_input, *d_output;

    checkCudaErrors(hipMalloc(&d_filter, filter->size() * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_input, input->size() * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_output, output->size() * sizeof(float)));

    checkCudaErrors(hipMemcpy(d_input, input->getData(), input->size() * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_filter, filter->getData(), filter->size() * sizeof(float), hipMemcpyHostToDevice));

    return {d_filter, d_input, d_output};
}

bool performComputationalFeasibility(std::tuple<int, int> sharedMemoryParams, std::tuple<int*, dim3> gridSizeParams,
                                     std::tuple<int*, dim3> blockSizeParams)
{
    auto [sharedMax, sharedAmount] = sharedMemoryParams;
    auto [gridMax, gridAmount] = gridSizeParams;
    auto [blockMax, blockAmount] = blockSizeParams;

    // shared memory check
    if (sharedMax < sharedAmount)
    {
        std::cout << "[ERROR]:: shared memory constraint violation\n";
        std::cout << "          * [max]\t!<\t[yours]\n";
        std::cout << "          * " << sharedMax << " !< " << sharedAmount << "\n";
        return false;
    }

    if (gridMax[0] < gridAmount.x * blockAmount.x || gridMax[1] < gridAmount.y * blockAmount.y || gridMax[2] <
        gridAmount.z * blockAmount.z)
    {
        std::cout << "[ERROR]:: grid size constraint violation\n";
        std::cout << "          * [max]\t!<\t[yours]\n";
        std::cout << "          * " << gridMax[0] << " !< " << gridAmount.x * blockAmount.x << "\n";
        std::cout << "          * " << gridMax[1] << " !< " << gridAmount.y * blockAmount.y << "\n";
        std::cout << "          * " << gridMax[2] << " !< " << gridAmount.z * blockAmount.z << "\n";
        return false;
    }

    if (blockMax[0] < blockAmount.x || blockMax[1] < blockAmount.y || blockMax[2] < blockAmount.z)
    {
        std::cout << "[ERROR]:: block size constraint violation\n";
        std::cout << "          * [max]\t!<\t[yours]\n";
        std::cout << "          * " << blockMax[0] << " !< " << blockAmount.x << "\n";
        std::cout << "          * " << blockMax[1] << " !< " << blockAmount.y << "\n";
        std::cout << "          * " << blockMax[2] << " !< " << blockAmount.z << "\n";

        return false;
    }

    return true;
}

std::variant<bool, std::tuple<dim3, dim3, std::pair<int, int>, std::pair<int, int>, int>> runSizeSetup(
    int width, int height, int convolutionType)
{
    auto inputParams = std::make_pair(width, height);
    auto filterParams = std::make_pair(FILTER_SIZE, FILTER_RADIUS);

    auto [gridSize, blockDim] = setSizeAndGrid(convolutionType, inputParams);

    int sharedMemSize = (blockDim.x + 2 * FILTER_RADIUS) * (blockDim.y + 2 * FILTER_RADIUS) * sizeof(float);

    auto [sharedMemConstraint, gridSizeConstraint, blockSizeConstraint] = getDeviceConstraints();

    std::cout << "\n[SETUP]:: set block size :=(" << blockDim.x << ", " << blockDim.y << ", " << blockDim.z << ")";
    std::cout << "\n          max size:=(" << blockSizeConstraint[0] << ", " << blockSizeConstraint[1] << ", " <<
        blockSizeConstraint[2] << ")";

    std::cout << "\n[SETUP]:: set grid size :=(" << gridSize.x << ", " << gridSize.y << ", " << gridSize.z << ")";
    std::cout << "\n          max size:=(" << gridSizeConstraint[0] << ", " << gridSizeConstraint[1] << ", " <<
        gridSizeConstraint[2] << ")";

    std::cout << "\n[SETUP]:: set shared mem size:= " << sharedMemSize;
    std::cout << "\n          max shared mem:= " << sharedMemConstraint << std::endl;

    if (!performComputationalFeasibility({sharedMemConstraint, sharedMemSize},
                                         {gridSizeConstraint, gridSize},
                                         {blockSizeConstraint, blockDim}))
    {
        return false;
    }

    return std::make_tuple(gridSize, blockDim, inputParams, filterParams, sharedMemSize);
}

template <typename T>
float launchSingleKernel(std::variant<bool, T> outcome, float* d_input, float* d_filter, float* d_output)
{
    auto [gridSize, blockDim, inputParams, filterParams, sharedMemSize] = std::get<std::tuple<
        dim3, dim3, std::pair<int, int>, std::pair<int, int>, int>>(outcome);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // 3. kernel launch
    // Record the start time
    checkCudaErrors(hipEventRecord(start));

    singleKernelConvolution2D<<<gridSize, blockDim, sharedMemSize>>>(d_input, d_filter, d_output, inputParams,
                                                                     filterParams);

    // 4. device synch & mem copy backwards
    checkCudaErrors(hipDeviceSynchronize());

    // Record the stop time
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

    // Calculate and display elapsed time
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    // Cleanup timing events
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}

float launchStream(float* d_input, float* d_filter, float* d_output, std::pair<int, int> inputSize)
{
    int tileWidth = TILE_WIDTH;
    int tileHeight = TILE_HEIGHT;

    int inputWidth = inputSize.first;
    int inputHeight = inputSize.second;

    int numTilesX = (inputWidth + tileWidth - 1) / tileWidth;
    int numTilesY = (inputHeight + tileHeight - 1) / tileHeight;

    int numStreams = 4;
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
    }

    int streamIdx = 0;
    int sharedMemSize = (tileWidth + 2 * FILTER_RADIUS) * (tileHeight + 2 * FILTER_RADIUS) * sizeof(float);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    for (int ty = 0; ty < numTilesY; ++ty)
    {
        for (int tx = 0; tx < numTilesX; ++tx)
        {
            int startX = tx * tileWidth;
            int startY = ty * tileHeight;

            dim3 blockSize(16, 16, 1);
            dim3 gridSize((tileWidth + blockSize.x - 1) / blockSize.x, (tileHeight + blockSize.y - 1) / blockSize.y,1);

            multiKernelConvolution2D<<<gridSize, blockSize, sharedMemSize, streams[streamIdx]>>>(
                d_input, d_output, inputWidth, inputHeight, d_filter, FILTER_SIZE, startX, startY);

            streamIdx = (streamIdx + 1) % numStreams;
        }
    }

    // Synchronize all streams
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamSynchronize(streams[i]);
    }

    // Cleanup streams
    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamDestroy(streams[i]);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

int run_assignment_cuda(int argc, char** argv)
{
    auto [width, height, convolutionType] = parseInput(argc, argv);

    std::cout << "[RUNNING INIT]:: startup ...\n";
    auto [input, output, filter] = initArgs(width, height);
    std::cout << "[RUNNING INIT]:: done ...\n";

    assert(output->size() == input->size());

    // 1. memory allocation
    auto [d_filter, d_input, d_output] = allocateAndInitDeviceMemory(filter, input, output);

    // 2. gridsize & blocksize setup
    auto outcome = runSizeSetup(width, height, convolutionType);

    float milliseconds = 0.0f;

    if (std::holds_alternative<bool>(outcome))
    {
        std::cout << "[MODE]:: toggled stream mode\n";
        milliseconds = launchStream(d_input, d_filter, d_output, {width, height});
    }
    else
    {
        std::cout << "[MODE]:: toggled single kernel mode\n";
        milliseconds = launchSingleKernel(outcome, d_input, d_filter, d_output);
    }

    std::cout << "\nKernel execution time: " << milliseconds << " ms\n";

    // Copy result back to host
    checkCudaErrors(hipMemcpy(output->getData(), d_output, output->size() * sizeof(float),
                               hipMemcpyDeviceToHost));

    std::cout << "\n[OUTPUT]\n";
    printDynamicArray(output);

    // Cleanup and deallocate memory
    delete output;
    delete filter;
    delete input;

    hipFree(d_filter);
    hipFree(d_input);
    hipFree(d_output);

    return EXIT_SUCCESS;
}
