#include "hip/hip_runtime.h"
#include "cuda_implementation.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)

// Tiling parameters (adjust based on device capabilities)
#define TILE_WIDTH 32  // You can adjust this based on device capabilities

// CUDA kernel for 2D convolution
__global__ void convolution2D_kernel(float *d_input, float *d_output, float *d_filter,
                                      int width, int height, int filterSize) {
    __shared__ float shared_input[TILE_WIDTH + FILTER_SIZE - 1][TILE_WIDTH + FILTER_SIZE - 1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Calculate global row and column for the output
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // Load data into shared memory, with padding to handle filter overlap
    for (int i = ty; i < TILE_WIDTH + FILTER_SIZE - 1; i += blockDim.y) {
        for (int j = tx; j < TILE_WIDTH + FILTER_SIZE - 1; j += blockDim.x) {
            int input_row = row + i - FILTER_RADIUS;
            int input_col = col + j - FILTER_RADIUS;
            if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
                shared_input[i][j] = d_input[input_row * width + input_col];
            } else {
                shared_input[i][j] = 0.0f;  // Pad with zeros for out-of-bound accesses
            }
        }
    }

    __syncthreads();  // Synchronize threads to ensure shared memory is fully loaded

    // Perform the convolution if the current thread corresponds to a valid output pixel
    if (ty < TILE_WIDTH && tx < TILE_WIDTH && row < height && col < width) {
        float conv_result = 0.0f;
        for (int i = 0; i < filterSize; ++i) {
            for (int j = 0; j < filterSize; ++j) {
                conv_result += shared_input[ty + i][tx + j] * d_filter[i * filterSize + j];
            }
        }
        d_output[row * width + col] = conv_result;
    }
}

void convolution2D(float *h_input, float *h_output, float *h_filter,
                   int width, int height, int filterSize) {
    float *d_input, *d_output, *d_filter;
    size_t inputSize = width * height * sizeof(float);
    size_t filterSizeInBytes = filterSize * filterSize * sizeof(float);
    size_t outputSize = width * height * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_output, outputSize);
    hipMalloc(&d_filter, filterSizeInBytes);

    // Copy input data and filter to device
    hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, filterSizeInBytes, hipMemcpyHostToDevice);

    // Set up CUDA streams
    const int numStreams = 4; // Example: we can use 4 streams
    hipStream_t streams[numStreams];

    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Define block size and grid size
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);  // Each block processes a TILE_WIDTH x TILE_WIDTH region
    int gridX = (width + TILE_WIDTH - 1) / TILE_WIDTH;
    int gridY = (height + TILE_WIDTH - 1) / TILE_WIDTH;
    dim3 gridSize(gridX, gridY);

    // Launch kernels for multiple tiles in parallel using CUDA streams
    for (int streamIdx = 0; streamIdx < numStreams; ++streamIdx) {
        convolution2D_kernel<<<gridSize, blockSize, 0, streams[streamIdx]>>>(
            d_input, d_output, d_filter, width, height, filterSize);
    }

    // Synchronize to ensure kernel execution is complete before copying data back
    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    // Copy the result back to host
    hipMemcpy(h_output, d_output, outputSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_filter);

    // Destroy streams
    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
}

int main_test(int dim, float* input, float* filter) {
    // Example configuration
    int width = dim; // Example image width (64x64 for testing, scale up as needed)
    int height = dim; // Example image height (64x64)
    int filterSize = FILTER_SIZE;

    // Allocate host memory for input image, filter, and output
    float *h_input = input;
    float *h_output = new float[width * height];
    // float *h_filter = new float[filterSize * filterSize];
    float *h_filter = filter;

    // Perform 2D convolution
    convolution2D(h_input, h_output, h_filter, width, height, filterSize);

    // Example output (just print the first few values)
    std::cout << "Output (first 10 values):" << std::endl;
    std::cout << "> (input) [ ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_input[i] << " ";
    }
    std::cout << " ]\n";
    std::cout << "> (filter) [ ";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_filter[i] << " ";
    }
    std::cout << " ]\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    // delete[] h_input;
    delete[] h_output;
    // delete[] h_filter;

    return 0;
}