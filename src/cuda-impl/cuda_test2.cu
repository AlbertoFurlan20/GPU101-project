#include "hip/hip_runtime.h"
#include "cuda_implementation.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)

// Tiling parameters (adjust based on device capabilities)
#define TILE_WIDTH 32  // You can adjust this based on device capabilities

// Utility macro for 2D indexing
#define IDX_2D(x, y, width) ((y) * (width) + (x))

__global__ void convolution2D_basic(const float* input, const float* kernel, float* output,
                              std::pair<int, int> inputSize, std::pair<int, int> filterParams)
{
    extern __shared__ float sharedIndexes[];

    int kernelRadius = filterParams.second;
    int kernelSize = filterParams.first;
    int width = inputSize.first;
    int height = inputSize.second;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int sharedX = threadIdx.x + kernelRadius;
    int sharedY = threadIdx.y + kernelRadius;

    int inputIndex = IDX_2D(x, y, width);

    // Load data into shared memory
    if (x < width && y < height)
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + sharedX] = input[inputIndex];
    }
    else
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + sharedX] = 0.0f;
    }

    // Load halo regions
    if (threadIdx.x < kernelRadius)
    {
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + (sharedX - kernelRadius)] =
            (x >= kernelRadius) ? input[IDX_2D(x - kernelRadius, y, width)] : 0.0f;
        sharedIndexes[sharedY * (blockDim.x + 2 * kernelRadius) + (sharedX + blockDim.x)] =
            (x + blockDim.x < width) ? input[IDX_2D(x + blockDim.x, y, width)] : 0.0f;
    }

    if (threadIdx.y < kernelRadius)
    {
        sharedIndexes[(sharedY - kernelRadius) * (blockDim.x + 2 * kernelRadius) + sharedX] =
            (y >= kernelRadius) ? input[IDX_2D(x, y - kernelRadius, width)] : 0.0f;
        sharedIndexes[(sharedY + blockDim.y) * (blockDim.x + 2 * kernelRadius) + sharedX] =
            (y + blockDim.y < height) ? input[IDX_2D(x, y + blockDim.y, width)] : 0.0f;
    }

    __syncthreads();

    // Apply convolution
    if (x < width && y < height)
    {
        float partialResult = 0.0f;

        for (int idxY = 0; idxY < kernelSize; ++idxY)
        {
            for (int idxX = 0; idxX < kernelSize; ++idxX)
            {
                int sharedInputY = sharedY - kernelRadius + idxY;
                int sharedInputX = sharedX - kernelRadius + idxX;
                partialResult += kernel[idxY * kernelSize + idxX] *
                    sharedIndexes[sharedInputY * (blockDim.x + 2 * kernelRadius) + sharedInputX];
            }
        }

        output[IDX_2D(x, y, width)] = partialResult;
    }
}

// Version 1: Tiling Only
__global__ void convolution2D_tiling(const float* input, const float* kernel, float* output,
                                     std::pair<int, int> inputSize, std::pair<int, int> filterParams)
{
    extern __shared__ float sharedIndexes[];

    int kernelRadius = filterParams.second;
    int kernelSize = filterParams.first;
    int width = inputSize.first;
    int height = inputSize.second;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int sharedX = threadIdx.x + kernelRadius;
    int sharedY = threadIdx.y + kernelRadius;

    int sharedWidth = blockDim.x + 2 * kernelRadius;
    int sharedHeight = blockDim.y + 2 * kernelRadius;

    // Load data into shared memory, including halo regions
    for (int tileY = threadIdx.y; tileY < sharedHeight; tileY += blockDim.y)
    {
        for (int tileX = threadIdx.x; tileX < sharedWidth; tileX += blockDim.x)
        {
            int globalX = blockIdx.x * blockDim.x + tileX - kernelRadius;
            int globalY = blockIdx.y * blockDim.y + tileY - kernelRadius;

            sharedIndexes[tileY * sharedWidth + tileX] =
                (globalX >= 0 && globalX < width && globalY >= 0 && globalY < height)
                    ? input[IDX_2D(globalX, globalY, width)]
                    : 0.0f;
        }
    }

    __syncthreads();

    // Apply convolution
    if (x < width && y < height)
    {
        float partialResult = 0.0f;

        for (int idxY = 0; idxY < kernelSize; ++idxY)
        {
            for (int idxX = 0; idxX < kernelSize; ++idxX)
            {
                int sharedInputY = sharedY - kernelRadius + idxY;
                int sharedInputX = sharedX - kernelRadius + idxX;
                partialResult += kernel[idxY * kernelSize + idxX] *
                    sharedIndexes[sharedInputY * sharedWidth + sharedInputX];
            }
        }

        output[IDX_2D(x, y, width)] = partialResult;
    }
}

// Version 2: Streams Only
void launch_convolution2D_streams(const float* input, const float* kernel, float* output,
                                  std::pair<int, int> inputSize, std::pair<int, int> filterParams,
                                  int numStreams)
{
    int width = inputSize.first;
    int height = inputSize.second;

    int streamHeight = (height + numStreams - 1) / numStreams;
    hipStream_t* streams = new hipStream_t[numStreams];

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
        int startRow = i * streamHeight;
        int endRow = min((i + 1) * streamHeight, height);

        if (startRow >= height) break;

        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (endRow - startRow + blockDim.y - 1) / blockDim.y);

        convolution2D_basic<<<gridDim, blockDim, (blockDim.x + 2 * filterParams.second) * (blockDim.y + 2 * filterParams.
            second) * sizeof(float), streams[i]>>>(
            input + startRow * width, kernel, output + startRow * width,
            {width, endRow - startRow}, filterParams);
    }

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    delete[] streams;
}

// Version 3: Tiling + Streams
void launch_convolution2D_tiling_streams(const float* input, const float* kernel, float* output,
                                         std::pair<int, int> inputSize, std::pair<int, int> filterParams,
                                         int numStreams)
{
    int width = inputSize.first;
    int height = inputSize.second;

    int streamHeight = (height + numStreams - 1) / numStreams;
    hipStream_t* streams = new hipStream_t[numStreams];

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
        int startRow = i * streamHeight;
        int endRow = min((i + 1) * streamHeight, height);

        if (startRow >= height) break;

        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (endRow - startRow + blockDim.y - 1) / blockDim.y);

        convolution2D_tiling<<<gridDim, blockDim, (blockDim.x + 2 * filterParams.second) * (blockDim.y + 2 *
            filterParams.second) * sizeof(float), streams[i]>>>(
            input + startRow * width, kernel, output + startRow * width,
            {width, endRow - startRow}, filterParams);
    }

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    delete[] streams;
}

int main_test2(int dim, float* input, float* filter)
{
    // Example configuration
    int width = dim;
    int height = dim;
    int filterSize = FILTER_SIZE;

    // Allocate host memory for input image, filter, and output
    float* h_input = input;
    float* h_filter = filter;
    float* h_output_basic = new float[width * height];
    float* h_output_tiling = new float[width * height];
    float* h_output_streams = new float[width * height];
    float* h_output_both = new float[width * height];

    // Allocate device memory
    float *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, width * height * sizeof(float));
    hipMalloc(&d_filter, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMalloc(&d_output, width * height * sizeof(float));

    hipMemcpy(d_input, h_input, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, FILTER_SIZE * FILTER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Kernel execution parameters
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    size_t sharedMemorySize = (blockDim.x + 2 * FILTER_RADIUS) * (blockDim.y + 2 * FILTER_RADIUS) * sizeof(float);

    // Run kernels
    convolution2D_basic<<<gridDim, blockDim, sharedMemorySize>>>(d_input, d_filter, d_output, {width, height}, {FILTER_SIZE, FILTER_RADIUS});
    hipMemcpy(h_output_basic, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    convolution2D_tiling<<<gridDim, blockDim, sharedMemorySize>>>(d_input, d_filter, d_output, {width, height}, {FILTER_SIZE, FILTER_RADIUS});
    hipMemcpy(h_output_tiling, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    launch_convolution2D_streams(d_input, d_filter, d_output, {width, height}, {FILTER_SIZE, FILTER_RADIUS}, 4);
    hipMemcpy(h_output_streams, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    launch_convolution2D_tiling_streams(d_input, d_filter, d_output, {width, height}, {FILTER_SIZE, FILTER_RADIUS}, 4);
    hipMemcpy(h_output_both, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    // Example output (just print the first few values)
    std::cout << "Output (first 10 values):" << std::endl;
    std::cout << "> (input) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_input[i] << " ";
    std::cout << "]\n";
    std::cout << "> (filter) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_filter[i] << " ";
    std::cout << "]\n";
    std::cout << "> (basic) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_output_basic[i] << " ";
    std::cout << "]\n";
    std::cout << "> (out-tiling) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_output_tiling[i] << " ";
    std::cout << "]\n";
    std::cout << "> (out-streams) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_output_streams[i] << " ";
    std::cout << "]\n";
    std::cout << "> (out-both) [ ";
    for (int i = 0; i < 10; ++i) std::cout << h_output_both[i] << " ";
    std::cout << "]\n";
    std::cout << std::endl;

    // Clean up
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    delete[] h_output_basic;
    delete[] h_output_tiling;
    delete[] h_output_streams;
    delete[] h_output_both;

    return 0;
}
